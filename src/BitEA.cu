#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
// #include <sys/time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <chrono>
#include <thrust/copy.h>

#include "BitEA.h"
#include "stdgraph.h"

#define CUDA_CHECK(call)                                                                                \
    {                                                                                                   \
        hipError_t err = call;                                                                         \
        if (err != hipSuccess)                                                                         \
        {                                                                                               \
            fprintf(stderr, "CUDA Error: %s, at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(1);                                                                                    \
        }                                                                                               \
    }

#define MEMORY_EXISTS(ptr) (ptr != nullptr && ptr != NULL)

#define DELETE_ARRAY(ptr)   \
    if (MEMORY_EXISTS(ptr)) \
    {                       \
        delete[] ptr;       \
        ptr = nullptr;      \
    }

#define DELETE_ARRAY_PTR(ptr)                                      \
    if (MEMORY_EXISTS(ptr))                                        \
    {                                                              \
        printf("Deleting %p at %s:%d\n", ptr, __FILE__, __LINE__); \
        free(ptr);                                                 \
        ptr = nullptr;                                             \
    }

// remaining gen count for each thread (multi-thread safe)
__device__ int remaining_gen_count;

__device__ int best_i_global = 0;
__device__ int best_i_lock = 0;

__device__ GpuTimer *gpuTimer;
__device__ float elapsed_time = 0.0;

__device__ hiprandState *curand_state;

__global__ void setup_kernel(unsigned long seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x; // Unique thread ID
    hiprand_init(seed + id, id, 0, &curand_state[id]);
}

__device__ inline int __rand()
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int value = hiprand(&curand_state[id]);
    if (value < 0)
        return -value;
    return value;
}

__device__ int lock_pair(int p1, int p2, int *mutexes, int population_size)
{
    // Ensure indices are in bounds
    if (p1 >= population_size || p2 >= population_size || p1 < 0 || p2 < 0)
    {
        return 0;
    }
    int combined = min(p1, p2) * population_size + max(p1, p2);
    return atomicCAS(&mutexes[combined], 0, 1) == 0;
}

__device__ void unlock_pair(int p1, int p2, int *mutexes, int population_size)
{
    int combined = min(p1, p2) * population_size + max(p1, p2);
    atomicExch(&mutexes[combined], 0);
}

__device__ int __popcountl(uint64_t n)
{
    return __popcll(n);
}

__device__ int __count_conflicts(int graph_size, const block_t *color, const block_t *edges, int *conflict_count)
{
    int i, j, total_conflicts = 0;
    for (i = 0; i < graph_size; i++)
    {
        if (CHECK_COLOR(color, i))
        {
            conflict_count[i] = 0;
            for (j = 0; j < TOTAL_BLOCK_NUM(graph_size); j++)
                conflict_count[i] += __popcountl(color[j] & edges[i * TOTAL_BLOCK_NUM(graph_size) + j]);
            total_conflicts += conflict_count[i];
        }
    }

    return total_conflicts / 2;
}

__device__ void fix_conflicts(int graph_size, const block_t *edges, const int *weights, int *conflict_count, int *total_conflicts, block_t *color, block_t *pool, int *pool_total)
{
    // Keep removing problematic vertices until all conflicts are gone.
    int i, worst_vert = 0, vert_block;
    block_t vert_mask;
    while (*total_conflicts > 0)
    {
        // Find the vertex with the most conflicts.
        for (i = 0; i < graph_size; i++)
        {
            if (CHECK_COLOR(color, i) &&
                (conflict_count[worst_vert] < conflict_count[i] ||
                 (conflict_count[worst_vert] == conflict_count[i] &&
                  (weights[worst_vert] > weights[i] || (weights[worst_vert] == weights[i] && __rand() % 2)))))
            {
                worst_vert = i;
            }
        }

        // Update other conflict counters.
        vert_mask = MASK(worst_vert);
        vert_block = BLOCK_INDEX(worst_vert);
        for (i = 0; i < graph_size; i++)
            if (CHECK_COLOR(color, i) && (edges[i * TOTAL_BLOCK_NUM(graph_size) + vert_block] & vert_mask))
                conflict_count[i]--;

        // Remove the vertex.
        color[vert_block] &= ~vert_mask;
        pool[vert_block] |= vert_mask;
        (*pool_total)++;

        // Update the total number of conflicts.
        (*total_conflicts) -= conflict_count[worst_vert];
        conflict_count[worst_vert] = 0;
    }
}

__device__ void merge_and_fix(int graph_size, const block_t *edges, const int *weights, const block_t **parent_color, block_t *child_color, block_t *pool, int *pool_count, block_t *used_vertex_list, int *used_vertex_count)
{
    // Merge the two colors
    int temp_v_count = 0;
    if (parent_color[0] != nullptr && parent_color[1] != nullptr)
        for (int i = 0; i < (TOTAL_BLOCK_NUM(graph_size)); i++)
        {
            child_color[i] = ((parent_color[0][i] | parent_color[1][i]) & ~(used_vertex_list[i]));
            temp_v_count += __popcountl(child_color[i]);
        }

    else if (parent_color[0] != nullptr)
        for (int i = 0; i < (TOTAL_BLOCK_NUM(graph_size)); i++)
        {
            child_color[i] = (parent_color[0][i] & ~(used_vertex_list[i]));
            temp_v_count += __popcountl(child_color[i]);
        }

    else if (parent_color[1] != nullptr)
        for (int i = 0; i < (TOTAL_BLOCK_NUM(graph_size)); i++)
        {
            child_color[i] = (parent_color[1][i] & ~(used_vertex_list[i]));
            temp_v_count += __popcountl(child_color[i]);
        }

    (*used_vertex_count) += temp_v_count;

    // Merge the pool with the new color
    for (int i = 0; i < (TOTAL_BLOCK_NUM(graph_size)); i++)
    {
        child_color[i] |= pool[i];
        used_vertex_list[i] |= child_color[i];
    }

    // memset(pool, 0, (TOTAL_BLOCK_NUM(graph_size)) * sizeof(block_t));
    for (int i = 0; i < TOTAL_BLOCK_NUM(graph_size); i++)
        pool[i] = 0;
    (*pool_count) = 0;

    int *conflict_count = (int *)malloc(graph_size * sizeof(int));
    for (int i = 0; i < graph_size; i++)
        conflict_count[i] = 0;

    // Count conflicts.
    int total_conflicts = __count_conflicts(graph_size, child_color, edges, conflict_count);

    // Fix the conflicts.
    fix_conflicts(graph_size, edges, weights, conflict_count, &total_conflicts, child_color, pool, pool_count);

    free(conflict_count);
}

__device__ void search_back(int graph_size, const block_t *edges, const int *weights, block_t *child, int color_count, block_t *pool, int *pool_count)
{
    int conflict_count, last_conflict, last_conflict_block = 0;
    block_t i_mask, temp_mask, last_conflict_mask = 0;
    int i, j, k, i_block;

    // Search back and try placing vertices from the pool in previous colors.
    for (i = 0; i < graph_size && (*pool_count) > 0; i++)
    {
        i_block = BLOCK_INDEX(i);
        i_mask = MASK(i);

        // Check if the vertex is in the pool.
        if (pool[i_block] & i_mask)
        {
            // Loop through every previous color.
            for (j = 0; j < color_count; j++)
            {
                // Count the possible conflicts in this color.
                conflict_count = 0;
                for (k = 0; k < TOTAL_BLOCK_NUM(graph_size); k++)
                {
                    temp_mask = child[j * TOTAL_BLOCK_NUM(graph_size) + k] & edges[i * TOTAL_BLOCK_NUM(graph_size) + k];
                    if (temp_mask)
                    {
                        conflict_count += __popcountl(temp_mask);
                        if (conflict_count > 1)
                            break;
                        last_conflict = sizeof(block_t) * 8 * (k + 1) - 1 - __clzll(temp_mask);
                        last_conflict_mask = temp_mask;
                        last_conflict_block = k;
                    }
                }

                // Place immediately if there are no conflicts.
                if (conflict_count == 0)
                {
                    child[j * TOTAL_BLOCK_NUM(graph_size) + i_block] |= i_mask;
                    pool[i_block] &= ~i_mask;
                    (*pool_count)--;
                    break;

                    // If only 1 conflict exists and its weight is smaller
                    // than that of the vertex in question, replace it.
                }
                else if (conflict_count == 1 && weights[last_conflict] < weights[i])
                {
                    child[j * TOTAL_BLOCK_NUM(graph_size) + i_block] |= i_mask;
                    pool[i_block] &= ~i_mask;

                    child[j * TOTAL_BLOCK_NUM(graph_size) + last_conflict_block] &= ~last_conflict_mask;
                    pool[last_conflict_block] |= last_conflict_mask;
                    break;
                }
            }
        }
    }
}

__device__ void local_search(int graph_size, const block_t *edges, const int *weights, block_t *child, int color_count, block_t *pool, int *pool_count)
{
    int i, j, k, h, i_block;
    block_t i_mask, temp_mask;
    int competition;
    int conflict_count;
    block_t *conflict_array = (block_t *)malloc(TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));
    for (i = 0; i < TOTAL_BLOCK_NUM(graph_size); i++)
        conflict_array[i] = 0;

    // Search back and try placing vertices from the pool in the colors.
    for (i = 0; i < graph_size && (*pool_count) > 0; i++)
    {
        i_block = BLOCK_INDEX(i);
        i_mask = MASK(i);

        // Check if the vertex is in the pool.
        if (pool[i_block] & i_mask)
        {
            // Loop through every color.
            for (j = 0; j < color_count; j++)
            {
                // Count conflicts and calculate competition
                conflict_count = 0;
                competition = 0;
                for (k = 0; k < TOTAL_BLOCK_NUM(graph_size); k++)
                {
                    conflict_array[k] = edges[i * TOTAL_BLOCK_NUM(graph_size) + k] & child[j * TOTAL_BLOCK_NUM(graph_size) + k];
                    if (conflict_array[k])
                    {
                        temp_mask = conflict_array[k];
                        conflict_count += __popcountl(temp_mask);
                        for (h = 0; h < sizeof(block_t) * 8; h++)
                            if ((temp_mask >> h) & (block_t)1)
                                competition += weights[k * 8 * sizeof(block_t) + h];
                    }
                }

                // Place immediately if there are no conflicts.
                if (competition == 0)
                {
                    child[j * TOTAL_BLOCK_NUM(graph_size) + i_block] |= i_mask;
                    pool[i_block] &= ~i_mask;
                    (*pool_count) += conflict_count - 1;
                    break;

                    /**
                     * If the total competition is smaller than the weight
                     * of the vertex in question, move all the conflicts to the
                     * pool, and place the vertex in the color.
                     */
                }
                else if (competition < weights[i])
                {
                    for (k = 0; k < TOTAL_BLOCK_NUM(graph_size); k++)
                    {
                        child[j * TOTAL_BLOCK_NUM(graph_size) + k] &= ~conflict_array[k];
                        pool[k] |= conflict_array[k];
                    }

                    child[j * TOTAL_BLOCK_NUM(graph_size) + i_block] |= i_mask;
                    pool[i_block] &= ~i_mask;
                    (*pool_count) += conflict_count - 1;
                    break;
                }
            }
        }
    }

    free(conflict_array);
}

__device__ int get_rand_color(int max_color_num, int colors_used, block_t used_color_list[])
{
    // There are no available colors.
    if (colors_used >= max_color_num)
    {
        return -1;

        // There are only 2 colors available, search for them linearly.
    }
    else if (colors_used > max_color_num - 2)
    {
        for (int i = 0; i < max_color_num; i++)
        {
            if (!(used_color_list[BLOCK_INDEX(i)] & MASK(i)))
            {
                used_color_list[BLOCK_INDEX(i)] |= MASK(i);
                return i;
            }
        }
    }

    // Randomly try to select an available color.
    int temp;
    while (1)
    {
        temp = __rand() % max_color_num;
        if (!(used_color_list[BLOCK_INDEX(temp)] & MASK(temp)))
        {
            used_color_list[BLOCK_INDEX(temp)] |= MASK(temp);
            return temp;
        }
    }
}

__device__ int crossover(int graph_size, const block_t *edges, const int *weights, int color_num1, int color_num2, const block_t *parent1, const block_t *parent2,
                         int target_color_count, block_t *child, int *child_color_count, int *uncolored)
{
    // max number of colors of the two parents.
    int max_color_num = color_num1 > color_num2 ? color_num1 : color_num2;

    // list of used colors in the parents.
    block_t **used_color_list = (block_t **)malloc(2 * sizeof(block_t *));
    used_color_list[0] = (block_t *)malloc(TOTAL_BLOCK_NUM(max_color_num) * sizeof(block_t));
    used_color_list[1] = (block_t *)malloc(TOTAL_BLOCK_NUM(max_color_num) * sizeof(block_t));
    for (int i = 0; i < TOTAL_BLOCK_NUM(max_color_num); i++)
    {
        used_color_list[0][i] = 0;
        used_color_list[1][i] = 0;
    }

    // list of used vertices in the parents.
    block_t *used_vertex_list = (block_t *)malloc(TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));
    for (int i = 0; i < TOTAL_BLOCK_NUM(graph_size); i++)
    {
        used_vertex_list[i] = 0;
    }
    int used_vertex_count = 0;

    // Pool.
    block_t *pool = (block_t *)malloc(TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));
    for (int i = 0; i < TOTAL_BLOCK_NUM(graph_size); i++)
    {
        pool[i] = 0;
    }
    int pool_count = 0;

    int color1, color2, last_color = 0;
    int i, j;
    const block_t *chosen_parent_colors[2];
    for (i = 0; i < target_color_count; i++)
    {
        // The child still has vertices that weren't used.
        if (used_vertex_count < graph_size)
        {
            // Pick 2 random colors.
            color1 = get_rand_color(color_num1, i, used_color_list[0]);
            color2 = get_rand_color(color_num2, i, used_color_list[1]);
            chosen_parent_colors[0] = color1 == -1 ? nullptr : &parent1[color1 * TOTAL_BLOCK_NUM(graph_size)];
            chosen_parent_colors[1] = color2 == -1 ? nullptr : &parent2[color2 * TOTAL_BLOCK_NUM(graph_size)];

            merge_and_fix(graph_size, edges, weights, chosen_parent_colors, &child[i * TOTAL_BLOCK_NUM(graph_size)], pool, &pool_count, used_vertex_list, &used_vertex_count);

            // If all of the vertices were used and the pool is empty, exit the loop.
        }
        else if (pool_count == 0)
        {
            break;
        }

        search_back(graph_size, edges, weights, child, i, pool, &pool_count);
    }

    // Record the last color.
    last_color = i;

    // If not all the vertices were visited, drop them in the pool.
    if (used_vertex_count < graph_size)
    {
        for (j = 0; j < TOTAL_BLOCK_NUM(graph_size); j++)
            pool[j] |= ~used_vertex_list[j];
        pool[TOTAL_BLOCK_NUM(graph_size)] &= ((0xFFFFFFFFFFFFFFFF) >> (TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t) * 8 - graph_size));

        pool_count += (graph_size - used_vertex_count);
        used_vertex_count = graph_size;
        for (i = 0; i < TOTAL_BLOCK_NUM(graph_size); i++)
            used_vertex_list[i] = 0xFF;
    }

    local_search(graph_size, edges, weights, child, target_color_count, pool, &pool_count);

    // If the pool is not empty, randomly allocate the remaining vertices in the colors.
    int fitness = 0, temp_block;
    block_t temp_mask;
    if (pool_count > 0)
    {
        int color_num;
        for (i = 0; i < graph_size; i++)
        {
            temp_block = BLOCK_INDEX(i);
            temp_mask = MASK(i);
            if (pool[temp_block] & temp_mask)
            {
                color_num = __rand() % target_color_count;
                child[color_num * TOTAL_BLOCK_NUM(graph_size) + temp_block] |= temp_mask;

                if (color_num + 1 > last_color)
                    last_color = color_num + 1;

                fitness += weights[i];
            }
        }
    }
    else
    {
        fitness = 0;
    }

    free(used_color_list[0]);
    free(used_color_list[1]);
    free(used_color_list);
    free(used_vertex_list);
    free(pool);

    *uncolored = pool_count;
    *child_color_count = last_color;
    return fitness;
}

__global__ void d_BitEA(int graph_size, block_t **population, block_t **children, int *color_count, int *uncolored, int *fitness, const block_t *edges, int *weights, int population_size,
                        int base_color_count, block_t *best_solution, int *best_fitness, float *best_solution_time, int *uncolored_num, int *mutexes, int *result)
{
    // Get the thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    bool has_printed = false;
    int target_color = base_color_count;
    int temp_uncolored = 0;
    int child_colors = 0, temp_fitness = 0;
    int bad_parent = 0;

    long long start = 0;
    auto end = clock64();
    auto best_end = clock64();

    while (atomicSub(&remaining_gen_count, 1) > 0)
    {
        int parent1_locked = 0;
        int parent2_locked = 0;
        int parent1 = -1, parent2 = -1;

        // Select first parent
        // In d_BitEA kernel
        do
        {
            // Reset parents if invalid
            if (parent1 >= population_size)
                parent1 = -1;
            if (parent2 >= population_size)
                parent2 = -1;

            // Select first parent
            if (parent1 == -1)
            {
                parent1 = __rand() % population_size;
            }
            else
            {
                parent1 = (parent1 + 1) % population_size;
            }

            // Select second parent
            if (parent2 == -1)
            {
                do
                {
                    parent2 = __rand() % population_size;
                } while (parent2 == parent1);
            }
            else
            {
                do
                {
                    parent2 = (parent2 + 1) % population_size;
                } while (parent2 == parent1);
            }

            // Try to lock both parents
            if (lock_pair(parent1, parent2, mutexes, population_size))
            {
                parent1_locked = 1;
                parent2_locked = 1;
            }
            else
            {
                parent1_locked = 0;
                parent2_locked = 0;
            }

        } while (!parent1_locked || !parent2_locked);

        for (int i = 0; i < base_color_count * TOTAL_BLOCK_NUM(graph_size); i++)
        {
            children[id][i] = 0;
        }

        temp_fitness = crossover(graph_size, edges, weights, color_count[parent1], color_count[parent2], population[parent1], population[parent2], target_color, children[id], &child_colors, &temp_uncolored);

        // Choose the bad parent.
        if (fitness[parent1] <= fitness[parent2] && color_count[parent1] <= color_count[parent2])
            bad_parent = parent2;
        else
            bad_parent = parent1;

        // Replace the bad parent if needed.
        if (child_colors <= color_count[bad_parent] && temp_fitness <= fitness[bad_parent])
        {
            // Copy child to bad_parent both memory is in device
            // memmove(population[bad_parent], child, (TOTAL_BLOCK_NUM(graph_size))*base_color_count*sizeof(block_t));
            for (int i = 0; i < base_color_count * TOTAL_BLOCK_NUM(graph_size); i++)
            {
                population[bad_parent][i] = children[id][i];
            }
            atomicExch(&color_count[bad_parent], child_colors);
            atomicExch(&fitness[bad_parent], temp_fitness);
            atomicExch(&uncolored[bad_parent], temp_uncolored);

            if (atomicCAS(&best_i_lock, 0, 1) == 0)
            {
                if (temp_fitness < fitness[best_i_global] || (temp_fitness == fitness[best_i_global] && child_colors < color_count[best_i_global]))
                {
                    best_i_global = bad_parent;
                    // if (temp_fitness == 0 && temp_uncolored == 0)
                    // {
                    //     atomicSub(&remaining_gen_count, remaining_gen_count);
                    // }
                }
                atomicExch(&best_i_lock, 0);
            }
        }

        // Make the target harder if it was found.
        if (temp_fitness == 0)
        {
            target_color = child_colors - 1;
        }

        unlock_pair(parent1, parent2, mutexes, population_size);

        end = clock64();
        if (id == 0 && (end - start) > 2500000000)
        {
            if (has_printed)
                printf("\033[A\033[K");
            has_printed = true;
            printf("Thread ID: %d | Generation: %d | Fitness: %d | Uncolored: %d | Color Count: %d\n", id, remaining_gen_count, temp_fitness, temp_uncolored, child_colors);
            start = end;
        }
    }

    // synchronize threads
    __syncthreads();

    // Copy the best solution to the global memory
    if (id == 0)
    {
        // memcpy(best_solution, population[best_i], base_color_count * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));
        for (int i = 0; i < base_color_count * TOTAL_BLOCK_NUM(graph_size); i++)
        {
            best_solution[i] = population[best_i_global][i];
        }
        *best_fitness = fitness[best_i_global];
        *uncolored_num = uncolored[best_i_global];
        *best_solution_time = remaining_gen_count;

        *result = color_count[best_i_global];

        if (has_printed)
            printf("\033[A\033[K");
    }
}

int BitEA(int graph_size, const block_t *edges, const int *weights, const int population_size, int base_color_count, int max_gen_num, block_t *best_solution, int *best_fitness, float *best_solution_time, int *uncolored_num)
{
    // int num_threads = 256; // Threads per block
    // int num_blocks = ((population_size) + num_threads - 1) / num_threads / 2;

    int num_blocks = population_size / 500;
    int num_threads = 250;
    if (num_blocks < 1)
    {
        num_blocks = 1;
        num_threads = population_size / 2;
    }

    int total_threads = num_threads * num_blocks;

    // // Create the random population.
    block_t **population = (block_t **)malloc(population_size * sizeof(block_t *));
    // void* original_address = population;  // Store the original address immediately
    // printf("Initial population address: %p\n", original_address);
    int *color_count = new int[population_size];
    int *uncolored = new int[population_size];
    int *fitness = new int[population_size];
    for (int i = 0; i < population_size; i++)
    {
        population[i] = (block_t *)malloc(base_color_count * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));
        // printf("Sub-array %d allocated: %p\n", i, (void*)population[i]);
        color_count[i] = base_color_count;
        uncolored[i] = base_color_count;
        fitness[i] = __INT_MAX__;
    }

    // populate the population with 0
    for (int i = 0; i < population_size; i++)
    {
        for (int j = 0; j < base_color_count * TOTAL_BLOCK_NUM(graph_size); j++)
        {
            population[i][j] = 0;
        }
    }

    pop_complex_random(graph_size, edges, weights, population_size, population, base_color_count);

    // Device memory
    block_t **d_population = nullptr;
    block_t **d_children = nullptr;
    block_t *d_edges = nullptr;
    int *d_weights = nullptr;
    int *d_color_count = nullptr;
    int *d_uncolored = nullptr;
    int *d_fitness = nullptr;

    // Best solution
    block_t *d_best_solution = nullptr;
    int *d_best_fitness = nullptr;
    float *d_total_execution_time = nullptr;
    int *d_best_color_count = nullptr;

    int *d_result = nullptr;

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMalloc(&d_population, population_size * sizeof(block_t *)));
    for (int i = 0; i < population_size; i++)
    {
        block_t *d_row;
        size_t row_size = base_color_count * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t);
        CUDA_CHECK(hipMalloc(&d_row, row_size);)
        for (int j = 0; j < base_color_count * TOTAL_BLOCK_NUM(graph_size); j++)
        {
            block_t value = population[i][j];
            CUDA_CHECK(hipMemcpy(&d_row[j], &value, sizeof(block_t), hipMemcpyHostToDevice);)
        }
        CUDA_CHECK(hipMemcpy(&d_population[i], &d_row, sizeof(block_t *), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_children, total_threads * sizeof(block_t *));)
    for (int i = 0; i < total_threads; i++)
    {
        block_t *d_row;
        size_t row_size = base_color_count * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t);
        CUDA_CHECK(hipMalloc(&d_row, row_size);)
        for (int j = 0; j < base_color_count * TOTAL_BLOCK_NUM(graph_size); j++)
        {
            block_t value = 0;
            CUDA_CHECK(hipMemcpy(&d_row[j], &value, sizeof(block_t), hipMemcpyHostToDevice);)
        }
        CUDA_CHECK(hipMemcpy(&d_children[i], &d_row, sizeof(block_t *), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_edges, graph_size * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));)
    for (int i = 0; i < graph_size * TOTAL_BLOCK_NUM(graph_size); i++)
    {
        block_t value = edges[i];
        CUDA_CHECK(hipMemcpy(&d_edges[i], &value, sizeof(block_t), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_weights, graph_size * sizeof(int));)
    for (int i = 0; i < graph_size; i++)
    {
        int value = weights[i];
        CUDA_CHECK(hipMemcpy(&d_weights[i], &value, sizeof(int), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_color_count, population_size * sizeof(int));)
    for (int i = 0; i < population_size; i++)
    {
        int value = color_count[i];
        CUDA_CHECK(hipMemcpy(&d_color_count[i], &value, sizeof(int), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_uncolored, population_size * sizeof(int));)
    for (int i = 0; i < population_size; i++)
    {
        int value = uncolored[i];
        CUDA_CHECK(hipMemcpy(&d_uncolored[i], &value, sizeof(int), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_fitness, population_size * sizeof(int));)
    for (int i = 0; i < population_size; i++)
    {
        int value = fitness[i];
        CUDA_CHECK(hipMemcpy(&d_fitness[i], &value, sizeof(int), hipMemcpyHostToDevice);)
    }

    CUDA_CHECK(hipMalloc(&d_best_solution, base_color_count * TOTAL_BLOCK_NUM((size_t)graph_size) * sizeof(block_t));)
    CUDA_CHECK(hipMalloc(&d_best_fitness, sizeof(int));)
    CUDA_CHECK(hipMalloc(&d_total_execution_time, sizeof(float));)
    CUDA_CHECK(hipMalloc(&d_best_color_count, sizeof(int));)
    CUDA_CHECK(hipMalloc(&d_result, sizeof(int));)

    // printf("Graph size: %d\n", graph_size);

    DELETE_ARRAY(color_count);
    // printf("Color count deleted\n");
    // fflush(stdout);
    DELETE_ARRAY(uncolored);
    // printf("Uncolored deleted\n");
    // fflush(stdout);
    DELETE_ARRAY(fitness);
    // printf("Fitness deleted\n");
    // fflush(stdout);

    int *d_mutexes;
    // When allocating mutexes array
    int mutex_array_size = population_size * population_size;
    hipMalloc(&d_mutexes, mutex_array_size * sizeof(int));
    hipMemset(d_mutexes, 0, mutex_array_size * sizeof(int));

    // Allocate memory for curand_state
    hiprandState *d_curand_state;
    CUDA_CHECK(hipMalloc(&d_curand_state, total_threads * sizeof(hiprandState)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(curand_state), &d_curand_state, sizeof(hiprandState *)));

    // Copy host remaining_gen_count to device
    int host_remaining_gen_count = max_gen_num;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(remaining_gen_count), &host_remaining_gen_count, sizeof(int)));

    // Initialize curand_state
    setup_kernel<<<num_blocks, num_threads>>>(time(nullptr));
    CUDA_CHECK(hipDeviceSynchronize());

#ifdef _WIN32
    auto start_time = std::chrono::high_resolution_clock::now();
#endif
#ifdef __linux__
    struct timespec start_time;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
#endif

    // Run the algorithm.
    d_BitEA<<<num_blocks, num_threads>>>(graph_size, d_population, d_children, d_color_count, d_uncolored, d_fitness, d_edges, d_weights, population_size, base_color_count, d_best_solution,
                                         d_best_fitness, d_total_execution_time, d_best_color_count, d_mutexes, d_result);

    // printf("Current population address: %p\n", (void*)population);
    // printf("Original population address: %p\n", original_address);
    // if (population != original_address) {
    //     printf("WARNING: Population pointer has changed!\n");
    // }

    // get last cuda error
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Wait for the kernel to finish.
    CUDA_CHECK(hipDeviceSynchronize());

#ifdef _WIN32
    auto end_time = std::chrono::high_resolution_clock::now();
    *best_solution_time = std::chrono::duration_cast<std::chrono::duration<float>>(end_time - start_time).count();
#endif
#ifdef __linux__
    struct timespec end_time;
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *best_solution_time = (end_time.tv_sec - start_time.tv_sec) + (end_time.tv_nsec - start_time.tv_nsec) / 1000000000.0;
#endif

    // Copy the results back to the host.
    // CUDA_CHECK(hipMemcpy(best_solution, d_best_solution, base_color_count * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t), hipMemcpyDeviceToHost);)
    block_t *d_best_solution_ptr = (block_t *)malloc(base_color_count * TOTAL_BLOCK_NUM(graph_size) * sizeof(block_t));
    for (int i = 0; i < base_color_count * TOTAL_BLOCK_NUM(graph_size); i++)
    {
        block_t value;
        CUDA_CHECK(hipMemcpy(&value, &d_best_solution[i], sizeof(block_t), hipMemcpyDeviceToHost);)
        d_best_solution_ptr[i] = value;
        best_solution[i] = d_best_solution_ptr[i];
    }
    CUDA_CHECK(hipMemcpy(best_fitness, d_best_fitness, sizeof(int), hipMemcpyDeviceToHost);)
    CUDA_CHECK(hipMemcpy(uncolored_num, d_best_color_count, sizeof(int), hipMemcpyDeviceToHost);)
    int result = -1;
    CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);)

    // Free device memory
    for (int i = 0; i < population_size; i++)
    {
        block_t *d_row;
        CUDA_CHECK(hipMemcpy(&d_row, &d_population[i], sizeof(block_t *), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_row));
    }
    CUDA_CHECK(hipFree(d_population));

    // Free children array
    for (int i = 0; i < total_threads; i++)
    {
        block_t *d_row;
        CUDA_CHECK(hipMemcpy(&d_row, &d_children[i], sizeof(block_t *), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_row));
    }
    CUDA_CHECK(hipFree(d_children));

    // Free other allocations
    CUDA_CHECK(hipFree(d_edges));
    CUDA_CHECK(hipFree(d_weights));
    CUDA_CHECK(hipFree(d_color_count));
    CUDA_CHECK(hipFree(d_uncolored));
    CUDA_CHECK(hipFree(d_fitness));
    CUDA_CHECK(hipFree(d_best_solution));
    CUDA_CHECK(hipFree(d_best_fitness));
    CUDA_CHECK(hipFree(d_total_execution_time));
    CUDA_CHECK(hipFree(d_best_color_count));
    CUDA_CHECK(hipFree(d_result));
    CUDA_CHECK(hipFree(d_mutexes));
    CUDA_CHECK(hipFree(d_curand_state));

    hipDeviceReset();

    hipError_t cuda_err = hipGetLastError();
    if (cuda_err != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cuda_err));
        exit(1);
    }

    // printf("Result: %d\n", result);

    // When freeing, only free sub-arrays but NOT the main array yet
    for (int i = 0; i < population_size; i++)
    {
        // printf("About to free sub-array %d: %p\n", i, (void*)population[i]);
        if (population[i])
        {
            free(population[i]);
            // printf("Freed sub-array %d\n", i);
            population[i] = nullptr;
        }
    }

    // Now try to read the main array to verify it's still valid
    // printf("Main array still at: %p\n", (void*)population);
    // for (int i = 0; i < population_size; i++) {
    //     printf("Sub-array pointer %d is now: %p\n", i, (void*)population[i]);
    // }

    // Finally try to free the main array
    // printf("About to free main array: %p\n", (void*)population);
    free(population); // Try direct free instead of DELETE_ARRAY_PTR
    population = nullptr;

    return result;
}